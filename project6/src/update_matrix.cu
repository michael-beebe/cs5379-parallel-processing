#include "hip/hip_runtime.h"
#include "update_matrix.h"

__global__ void update_matrix(int *D, int n) {
  int k = blockIdx.x; // Current phase based on block index
  int i = threadIdx.y + blockDim.y * blockIdx.y; // Row index
  int j = threadIdx.x + blockDim.x * blockIdx.z; // Column index

  if (i < n && j < n) {
    __shared__ int kRow[MATRIX_SIZE];
    __shared__ int kCol[MATRIX_SIZE];

    // Load the k-th row and column into shared memory
    if (threadIdx.x == 0 && i < n) kCol[i] = D[i * n + k];
    if (threadIdx.y == 0 && j < n) kRow[j] = D[k * n + j];

    __syncthreads(); // Ensure loading is complete

    // Update the matrix
    if (i != j) {
      atomicMin(&D[i * n + j], kCol[i] + kRow[j]);
    }
  }
}

void run_update_matrix(int *D, int n) {
  int *dev_D;

  // Allocate memory on the device
  hipMalloc((void**)&dev_D, n * n * sizeof(int));
  hipMemcpy(dev_D, D, n * n * sizeof(int), hipMemcpyHostToDevice);

  // Define grid and block sizes
  dim3 blocks(n, 1, 1); // One block per phase
  dim3 threadsPerBlock(THREADS_PER_BLOCK / n, n);

  // Launch the kernel
  for (int k = 0; k < n; ++k) {
    update_matrix<<<blocks, threadsPerBlock>>>(dev_D, n);
  }

  // Synchronize and copy back results
  hipDeviceSynchronize();
  hipMemcpy(D, dev_D, n * n * sizeof(int), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(dev_D);
}
