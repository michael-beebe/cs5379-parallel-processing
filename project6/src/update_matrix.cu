#include "hip/hip_runtime.h"
__global__ void update_matrix() {
  int k = blockIdx.x; // Current phase based on block index
  int i = threadIdx.y + blockDim.y * blockIdx.y; // Row index
  int j = threadIdx.x + blockDim.x * blockIdx.x; // Column index

  // Check bounds
  if (i < n && j < n) {
    // Copy the k-th row and column to shared memory
    __shared__ int hbuf[MAX_N];
    __shared__ int vbuf[MAX_N];

    if (threadIdx.x == 0) vbuf[i] = D[i * n + k];
    if (threadIdx.y == 0) hbuf[j] = D[k * n + j];

    __syncthreads(); // Ensure all threads have written to shared memory

    // Update the matrix D
    if (i != j) {
      D[i * n + j] = min(D[i * n + j], vbuf[i] + hbuf[j]);
    }
  }
}



void run_update_matrix(int *D, int n) {
  int *dev_D;

  // Allocate memory on the device
  hipMalloc((void**)&dev_D, n * n * sizeof(int));
  hipMemcpy(dev_D, D, n * n * sizeof(int), hipMemcpyHostToDevice);

  // Launch kernel with 32 blocks and 256 threads per block
  int numBlocks = 32;
  dim3 blocks(numBlocks, numBlocks);
  dim3 threadsPerBlock(THREADS_PER_BLOCK / numBlocks, THREADS_PER_BLOCK / numBlocks);
  updateMatrix<<<blocks, threadsPerBlock>>>(dev_D, n);

  // Copy result back to host
  hipMemcpy(D, dev_D, n * n * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_D);
}






// void run_update_matrix(int *D, int n) {
//   int *dev_D;
  
//   // Allocate memory on the device
//   hipMalloc((void**)&dev_D, n * n * sizeof(int));
//   hipMemcpy(dev_D, D, n * n * sizeof(int), hipMemcpyHostToDevice);

//   // Launch kernel
//   dim3 blocks(n, n / THREADS_PER_BLOCK); 
//   dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
//   update_matrix<<<blocks, threadsPerBlock>>>(dev_D, n);

//   // Copy result back to host
//   hipMemcpy(D, dev_D, n * n * sizeof(int), hipMemcpyDeviceToHost);
//   hipFree(dev_D);
// }