#include "hip/hip_runtime.h"
#include "update_matrix.h"

__global__ void update_matrix(int *D, int n) {
  int k = blockIdx.x; // Current phase based on block index
  int i = threadIdx.y + blockDim.y * blockIdx.y; // Row index
  int j = threadIdx.x + blockDim.x * blockIdx.z; // Column index

  // Check bounds
  if (i < n && j < n && k < n) {
    // Copy the k-th row and column to shared memory
    __shared__ int hbuf[MATRIX_SIZE];
    __shared__ int vbuf[MATRIX_SIZE];

    if (threadIdx.x == 0) vbuf[i] = D[i * n + k];
    if (threadIdx.y == 0) hbuf[j] = D[k * n + j];

    __syncthreads(); // Ensure all threads have written to shared memory

    // Update the matrix D
    if (i != j) {
      D[i * n + j] = min(D[i * n + j], vbuf[i] + hbuf[j]);
    }
  }
}

void run_update_matrix(int *D, int n) {
  int *dev_D;

  // Allocate memory on the device
  hipMalloc((void**)&dev_D, n * n * sizeof(int));
  hipMemcpy(dev_D, D, n * n * sizeof(int), hipMemcpyHostToDevice);

  // Calculate dimensions for blocks and threads
  dim3 blocks(BLOCKS_PER_GRID, BLOCKS_PER_GRID, n); // Using 3D grid for phases and matrix rows/columns
  dim3 threadsPerBlock((THREADS_PER_BLOCK / BLOCKS_PER_GRID), (THREADS_PER_BLOCK / BLOCKS_PER_GRID));

  // Launch kernel
  update_matrix<<<blocks, threadsPerBlock>>>(dev_D, n);

  // Synchronize device
  hipDeviceSynchronize();

  // Copy result back to host
  hipMemcpy(D, dev_D, n * n * sizeof(int), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(dev_D);
}
